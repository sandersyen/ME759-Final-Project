#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "clahe.cuh"

#define BIN_SIZE 101

__global__ void clahe(float* L, int width, int height, int threshold)
{
    __shared__ int bins[BIN_SIZE];
    __shared__ float cdf[BIN_SIZE];

    computeHistogram(L, width, height, bins);
    clipHistogram(bins, threshold);
    generateCdf(bins, cdf);
    mappingCdf(L, width, height, cdf);

} // <<<64, 1024>>>

__global__ void transformRgbToLab(unsigned char* pixels, int width, int height, float* L, float* A, float* B){
    // pixels: length = width * height * 3
    // L: length = width * height
    // A: length = width * height
    // B: length = width * height

    //https://stackoverflow.com/questions/49150250/convert-bgr-to-lab-without-opencv
    //http://www.easyrgb.com/en/math.php

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < width * height)
    {
        float r = (float)pixels[3*i]/255;
        float g = (float)pixels[3*i+1]/255;
        float b = (float)pixels[3*i+2]/255;

        r = (r > 0.04045) ? std::pow((r + 0.055) / 1.055, 2.4) : r / 12.92; r *= 100;
        g = (g > 0.04045) ? std::pow((g + 0.055) / 1.055, 2.4) : g / 12.92; g *= 100;
        b = (b > 0.04045) ? std::pow((b + 0.055) / 1.055, 2.4) : b / 12.92; b *= 100;

        // reference standard sRGB
        float x = (r * 0.4124 + g * 0.3576 + b * 0.1805) / 94.811;
        float y = (r * 0.2126 + g * 0.7152 + b * 0.0722) / 100.000;
        float z = (r * 0.0193 + g * 0.1192 + b * 0.9505) / 107.304;

        x = (x > 0.008856)? std::pow(x, 0.3333) : (7.787 * x) + 0.137931;
        y = (y > 0.008856)? std::pow(y, 0.3333) : (7.787 * y) + 0.137931;
        z = (z > 0.008856)? std::pow(z, 0.3333) : (7.787 * z) + 0.137931;

        L[i] = (116.0 * y) - 16.0;
        A[i] = 500.0 * (x - y);
        B[i] = 200.0 * (y - z);
    }
}
// https://github.com/berendeanicolae/ColorSpace/blob/master/src/Conversion.cpp

__global__ void transformLabToRgb(unsigned char* pixels, int width, int height, float* L, float* A, float* B)
{
    // Reference: https://stackoverflow.com/questions/7880264/convert-lab-color-to-rgb

    // pixels: length = width * height * 3
    // L: length = width * height
    // A: length = width * height
    // B: length = width * height
    int i = (threadIdx.x + blockDim.x * blockIdx.x);

    if (i < width * height)
    {
        float y = (L[i] + 16. ) / 116.;
        float x = A[i] / 500. + y;
        float z = y - B[i] / 200.;

        y = (pow(y, 3) > 0.008856) ? pow(y, 3) : (y - 16. / 116.) / 7.787;                      
        x = (pow(x, 3) > 0.008856) ? pow(x, 3) : (x - 16. / 116.) / 7.787;
        z = (pow(z,3) > 0.008856) ? pow(z,3) : (z - 16. / 116.) / 7.787;

        x = 95.047 * x / 100.;
        y = 100.000 * y / 100.;
        z = 108.883 * z / 100.;

        float r = x * 3.2406 + y * -1.5372 + z * -0.4986;
        float g = x * -0.9689 + y * 1.8758 + z * 0.0415;
        float b = x * 0.0557 + y * -0.2040 + z * 1.0570;

        r = (r > 0.0031308) ? 1.055 * pow(r , (1 / 2.4)) - 0.055 : 12.92 * r;
        g = (g > 0.0031308) ? 1.055 * pow(g , (1 / 2.4)) - 0.055 : 12.92 * g;
        b = (b > 0.0031308) ? 1.055 * pow( b , (1 / 2.4)) - 0.055 : 12.92 * b;
        
        pixels[3 * i] = (unsigned char)(r * 255.);
        pixels[3 * i + 1] = (unsigned char)(g * 255.);
        pixels[3 * i + 2] = (unsigned char)(b * 255.);
    }
}

// int block_dim = 32;
// dim3 dimBlock(block_dim, block_dim);
// dim3 dimGrid((width + dimBlock.x - 1)/dimBlock.x, (height+dimBlock.y -1)/dimBlock.y );
// kernel<<<dimGrid, dimBlock>>>();

// int block_size = blockDim.x;
// int row = threadIdx.y + blockIdx.y * block_size;
// int col = threadIdx.x + blockIdx.x * block_size;
// int i = row * width + col;

// called by kernel<<<dimGrid, dimBlock>>>()
__device__ void computeHistogram(float* L, int width, int height, int* bins)
{
    // L: length = width * height
    // bins: length = BIN_SIZE
    // int i = threadIdx.x + blockDim.x * blockIdx.x;
    int block_size = blockDim.x;
    int row = threadIdx.y + blockIdx.y * block_size;
    int col = threadIdx.x + blockIdx.x * block_size;
    int i = row * width + col;

    if (i < width * height)
    {
        int c = (int)L[i];
        atomicAdd(&bins[c], 1);
    }
}

// called by kernel<<<dimGrid, dimBlock>>>()
__device__ void clipHistogram(int* bins, int threshold)
{
    __shared__ int count_overlimit;
    // int i = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.x + threadIdx.y * blockDim.x;

    if(i == 0) count_overlimit=0;
    __syncthreads();

    if(i < BIN_SIZE)
    {
        if(bins[i] > threshold)
        {   
            atomicAdd(&count_overlimit, bins[i] - threshold);
            bins[i] = threshold;
        }
    }
    __syncthreads();
    if(i < BIN_SIZE) bins[i] = bins[i] + count_overlimit/BIN_SIZE + (i < count_overlimit%BIN_SIZE);
}

// called by kernel<<<dimGrid, dimBlock>>>()
__device__ void generateCdf(int* bins, float* cdf)
{
    // int i = threadIdx.x;
    int i = threadIdx.x + threadIdx.y * blockDim.x;
    // small array so here use sequential scan
    if(i == 0){
        for(int j = 1; j < BIN_SIZE; j++)
            bins[j]= bins[j]+bins[j-1];
    }
    __syncthreads();
    
    if( i < BIN_SIZE )
        cdf[i] = (float)bins[i]/(float)bins[BIN_SIZE-1];
}

__device__ void mappingCdf(float* L, int width, int height, float* cdf)
{
    // int i = threadIdx.x + blockDim.x * blockIdx.x;
    int block_size = blockDim.x;
    int row = threadIdx.y + blockIdx.y * block_size;
    int col = threadIdx.x + blockIdx.x * block_size;
    int i = row * width + col;
    int max = width * height;

    // average neighbor -> mapping
    if(i < max){

        int index = (int)L[i];
        int counter = 1;
        if ((threadIdx.y) != 0)
        {
            index += (int)L[i - width];
            ++counter;
        }

        if ((threadIdx.y + 1) < block_size && i + width < max)
        {
            index += (int)L[i + width];
            ++counter;
        }

        if ((threadIdx.x) != 0)
        {
            index += (int)L[i - 1];
            ++counter;
        }

        if ((threadIdx.x + 1) < block_size && i + 1 < max)
        {
            index += (int)L[i + 1];
            ++counter;
        }
        
        // int index = (int)L[i];
        float temp = cdf[index / counter] * 100;
        __syncthreads();

        L[i] = temp;
    }
}

