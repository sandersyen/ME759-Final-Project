#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "clahe.cuh"

#define BIN_SIZE 101

__global__ void clahe(){} // <<<64, 1024>>>

__global__ void transformRgbToLab(unsigned char* pixels, int width, int height, float* L, float* A, float* B){
    // pixels: length = width * height * 3
    // L: length = width * height
    // A: length = width * height
    // B: length = width * height

    //https://stackoverflow.com/questions/49150250/convert-bgr-to-lab-without-opencv
    //http://www.easyrgb.com/en/math.php

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < width * height)
    {
        float r = (float)pixels[3*i]/255;
        float g = (float)pixels[3*i+1]/255;
        float b = (float)pixels[3*i+2]/255;

        r = (r > 0.04045) ? std::pow((r + 0.055) / 1.055, 2.4) : r / 12.92; r *= 100;
        g = (g > 0.04045) ? std::pow((g + 0.055) / 1.055, 2.4) : g / 12.92; g *= 100;
        b = (b > 0.04045) ? std::pow((b + 0.055) / 1.055, 2.4) : b / 12.92; b *= 100;

        // reference standard sRGB
        float x = (r * 0.4124 + g * 0.3576 + b * 0.1805) / 94.811;
        float y = (r * 0.2126 + g * 0.7152 + b * 0.0722) / 100.000;
        float z = (r * 0.0193 + g * 0.1192 + b * 0.9505) / 107.304;

        x = (x > 0.008856)? std::pow(x, 0.3333) : (7.787 * x) + 0.137931;
        y = (y > 0.008856)? std::pow(y, 0.3333) : (7.787 * y) + 0.137931;
        z = (z > 0.008856)? std::pow(z, 0.3333) : (7.787 * z) + 0.137931;

        L[i] = (116.0 * y) - 16.0;
        A[i] = 500.0 * (x - y);
        B[i] = 200.0 * (y - z);
    }
}
// https://github.com/berendeanicolae/ColorSpace/blob/master/src/Conversion.cpp

__global__ void transformLabToRgb(unsigned char* pixels, int width, int height, float* L, float* A, float* B)
{
    // Reference: https://stackoverflow.com/questions/7880264/convert-lab-color-to-rgb

    // pixels: length = width * height * 3
    // L: length = width * height
    // A: length = width * height
    // B: length = width * height
    int i = (threadIdx.x + blockDim.x * blockIdx.x);

    if (i < width * height)
    {
        float y = (L[i] + 16. ) / 116.;
        float x = A[i] / 500. + y;
        float z = y - B[i] / 200.;

        y = (pow(y, 3) > 0.008856) ? pow(y, 3) : (y - 16. / 116.) / 7.787;                      
        x = (pow(x, 3) > 0.008856) ? pow(x, 3) : (x - 16. / 116.) / 7.787;
        z = (pow(z,3) > 0.008856) ? pow(z,3) : (z - 16. / 116.) / 7.787;

        x = 95.047 * x / 100.;
        y = 100.000 * y / 100.;
        z = 108.883 * z / 100.;

        float r = x * 3.2406 + y * -1.5372 + z * -0.4986;
        float g = x * -0.9689 + y * 1.8758 + z * 0.0415;
        float b = x * 0.0557 + y * -0.2040 + z * 1.0570;

        r = (r > 0.0031308) ? 1.055 * pow(r , (1 / 2.4)) - 0.055 : 12.92 * r;
        g = (g > 0.0031308) ? 1.055 * pow(g , (1 / 2.4)) - 0.055 : 12.92 * g;
        b = (b > 0.0031308) ? 1.055 * pow( b , (1 / 2.4)) - 0.055 : 12.92 * b;
        
        pixels[3 * i] = (char)(r * 255.);
        pixels[3 * i + 1] = (char)(g * 255.);
        pixels[3 * i + 2] = (char)(b * 255.);
    }
}

__global__ void computeHistogram(float* L, int width, int height, int* bins)
{
    // L: length = width * height
    // bins: length = BIN_SIZE
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < width * height)
    {
        int c = (int)L[i];
        atomicAdd(&bins[c], 1);
    }
}

__global__ void clipHistogram(int* bins, int threshold)
{
    __shared__ int count_overlimit;
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i == 0) count_overlimit=0;
    __syncthreads();

    if(i < BIN_SIZE)
    {
        if(bins[i] > threshold)
        {   
            atomicAdd(&count_overlimit, bins[i] - threshold);
            bins[i] = threshold;
        }
    }
    __syncthreads();
    if(i < BIN_SIZE) bins[i] = bins[i] + count_overlimit/BIN_SIZE + (i < count_overlimit%BIN_SIZE);
}

__global__ void generateCdf(int* bins, float* cdf)
{
    int i = threadIdx.x;
    // small array so here use sequential scan
    if(i==0){
        for(int j = 1; j < BIN_SIZE; j++)
            bins[j]= bins[j]+bins[j-1];
    }
    __syncthreads();
    
    if( i < BIN_SIZE )
        cdf[i] = (float)bins[i]/(float)bins[BIN_SIZE-1];
}

__global__ void mappingCdf(float* L, float* cdf)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // average neighbor -> mapping
    int index = (int)L[i];
    L[i] = cdf[index] * 100;
}

