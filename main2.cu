// UW ID: cchang253, syeh6
// Name: Chun-Ming Chang, Shang-Yen Yeh

#include <iostream>
#include <string>
#include "clahe2.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define BIN_SIZE 101

using namespace std;

int main(int argc, char *argv[])
{
    /*
        the way to run this program should be ./program_name [input_img_name] [output_img_name] [grid_size] [threshold]
    */
    if (argc < 3)
    {
        cout << "Didn't give enough arguments while calling CLAHE editor!\n";
    }
    else
    {
        try
        {
            string inputImg = argv[1];
            string outputImg = argv[2];
            int grid_size = stol(argv[3]);
            int threshold = stol(argv[4]);
            
            int width, height, channel;
            unsigned char* rgb_image = stbi_load(inputImg.c_str(), &width, &height, &channel, 3); // 3 means RGB
            int N = width * height;
            int threads_per_block = grid_size * grid_size;
            int num_block = ((width + grid_size - 1) / grid_size) * ((height + grid_size - 1) / grid_size);

            float* dL;
            float* dA;
            float* dB;
            unsigned char* dImg;

            hipMallocManaged((void **) &dImg, N*channel*sizeof(unsigned char));
            hipMallocManaged((void **) &dL, N * sizeof(float));
            hipMallocManaged((void **) &dA, N * sizeof(float));
            hipMallocManaged((void **) &dB, N * sizeof(float));

            hipMemcpy(dImg, rgb_image, N*channel*sizeof(unsigned char), hipMemcpyHostToDevice);
            hipMemset(dL, 0.0, N * sizeof(float));
            hipMemset(dA, 0.0, N * sizeof(float));
            hipMemset(dB, 0.0, N * sizeof(float));

            // Time the calculation actions exception for read and write image.
            hipEvent_t start;
            hipEvent_t stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);

            transformRgbToLab<<<num_block, threads_per_block>>>(dImg, width, height, dL, dA, dB);
            hipDeviceSynchronize();

            int block_dim = grid_size;
            dim3 dimBlock(block_dim, block_dim);
            dim3 dimGrid((width + dimBlock.x - 1)/dimBlock.x, (height+dimBlock.y -1)/dimBlock.y );

            float* dCdf;
            hipMallocManaged((void **) &dCdf, num_block * BIN_SIZE * sizeof(float));
            hipMemset(dCdf, 0.0, num_block * BIN_SIZE * sizeof(float));

            clahe<<<dimGrid, dimBlock>>>(dL, width, height, threshold, dCdf);
            hipDeviceSynchronize();
            
            pixelInterpolate<<<dimGrid, dimBlock>>>(dL, width, height, dCdf);
            hipDeviceSynchronize();

            transformLabToRgb<<<num_block, threads_per_block>>>(dImg, width, height, dL, dA, dB);
            hipDeviceSynchronize();

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            // Get the elapsed time in milliseconds
            float ms;
            hipEventElapsedTime(&ms, start, stop);
            cout << ms << endl;


            hipMemcpy(rgb_image, dImg, N*channel*sizeof(unsigned char), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            stbi_write_png(outputImg.c_str(), width, height, channel, rgb_image, width*3);

            stbi_image_free(rgb_image);

            hipFree(dL); hipFree(dA); hipFree(dB); hipFree(dImg);
        }
        catch (...)
        {
            cout << "Invalid argument!\n";
        }
    }

    cout << "\n";
    return 0;
}